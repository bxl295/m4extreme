#include "hip/hip_runtime.h"
/* -*- C++ -*- */

#include "powerDistanceCuda.h"
#include "../cuda/check.h"
#include "hip/hip_runtime.h"
#ifndef __CUDA_ARCH__
#include "gridCuda.h"
#include "ballsCuda.h"
#endif

#include <cassert>

namespace levelSet {


__global__
void
powerDistanceKernel(float* patch,
                    const float3 lowerCorner,
                    const float spacing,
                    const std::size_t numBalls,
                    const float4* ballsSquared) {
   // Calculate the Cartesian location of the grid point.
   const float3 p = {lowerCorner.x + spacing * threadIdx.x,
                     lowerCorner.y + spacing * threadIdx.y,
                     lowerCorner.z + spacing * threadIdx.z};
   // Initialize the grid value to infinity.
   float g = 1./0;
   // Find the minimum power distance over the set of balls.
   float d;
   float4 b;
   for (std::size_t i = 0; i != numBalls; ++i) {
      b = ballsSquared[i];
      d = (p.x - b.x) * (p.x - b.x) + (p.y - b.y) * (p.y - b.y) +
         (p.z - b.z) * (p.z - b.z) - b.w;
      if (d < g) {
         g = d;
      }
   }
   // Record the patch value. Convert the multi-index to a single offset.
   patch[threadIdx.x + threadIdx.y * blockDim.x +
         threadIdx.z * blockDim.x * blockDim.y] = g;
}


#ifndef __CUDA_ARCH__
// Translate the data to CUDA format and call powerDistanceKernel().
void
powerDistanceCuda(array::EquilateralArray<float, 3, PatchExtent>* patch,
                  const std::tr1::array<float, 3>& lowerCorner,
                  const float spacing,
                  const std::vector<geom::BallSquared<float, 3> >& balls) {
   // Allocate memory for the patch.
   float* patchDev;
   CUDA_CHECK(hipMalloc((void**)&patchDev, patch->size() * sizeof(float)));

   // Allocate memory for the balls and copy the memory.
   float4* ballsDev;
   CUDA_CHECK(hipMalloc((void**)&ballsDev, balls.size() * sizeof(float4)));
   {
      std::vector<float4> buffer(balls.size());
      for (std::size_t i = 0; i != balls.size(); ++i) {
         buffer[i].x = balls[i].center[0];
         buffer[i].y = balls[i].center[1];
         buffer[i].z = balls[i].center[2];
         buffer[i].w = balls[i].squaredRadius;
      }
      CUDA_CHECK(hipMemcpy(ballsDev, &buffer[0],
                            buffer.size() * sizeof(float4),
                            hipMemcpyHostToDevice));
   }

   const dim3 ThreadsPerBlock(PatchExtent, PatchExtent, PatchExtent);
   const float3 lower = {lowerCorner[0], lowerCorner[1], lowerCorner[2]};
   powerDistanceKernel<<<1,ThreadsPerBlock>>>
      (patchDev, lower, spacing, balls.size(), ballsDev);

   CUDA_CHECK(hipMemcpy(&(*patch)[0], patchDev, patch->size() * sizeof(float),
                         hipMemcpyDeviceToHost));
   CUDA_CHECK(hipFree(patchDev));
   CUDA_CHECK(hipFree(ballsDev));

#if 0
   for (std::size_t i = 0; i != patch->size(); ++i) {
      assert((*patch)[i] == 0);
   }
#else
   std::cout << *patch << '\n';
#endif
}
#endif


// Note: Storing ballsSquared in constant memory does not significantly affect
// performance.
__device__
void
powerDistanceKernel(float* patch,
                    const float3 lowerCorner,
                    const float spacing,
                    const float4* ballsSquared,
                    std::size_t numInfluencingBalls,
                    const std::size_t* ballIndices) {
   // Calculate the Cartesian location of the grid point.
   const float3 p = {lowerCorner.x + spacing * threadIdx.x,
                     lowerCorner.y + spacing * threadIdx.y,
                     lowerCorner.z + spacing * threadIdx.z};
   // Initialize the grid value to infinity.
   float g = 1./0;
   // Find the minimum power distance over the set of balls.
   float d;
#if 0
   float4 b;
   for (std::size_t i = 0; i != numInfluencingBalls; ++i) {
      b = ballsSquared[ballIndices[i]];
      d = (p.x - b.x) * (p.x - b.x) + (p.y - b.y) * (p.y - b.y) +
         (p.z - b.z) * (p.z - b.z) - b.w;
      if (d < g) {
         g = d;
      }
   }
   // Record the patch value. Convert the multi-index to a single offset.
   patch[threadIdx.x + threadIdx.y * blockDim.x +
         threadIdx.z * blockDim.x * blockDim.y] = g;
#else
   // Using shared memory improves performance on the 9600M GT, but not on
   // the 9400M.
   __shared__ float4 b[32];
   // Convert the multi-index to a single offset.
   const std::size_t tid = threadIdx.x + threadIdx.y * blockDim.x +
      threadIdx.z * blockDim.x * blockDim.y;
   std::size_t block;
   while (numInfluencingBalls != 0) {
      block = 32;
      if (numInfluencingBalls < 32) {
         block = numInfluencingBalls;
      }
      // Load the balls into shared memory.
      if (tid < block) {
         // Even though these are uncoalesced, this is an insignificant cost.
         b[tid] = ballsSquared[ballIndices[tid]];
      }
      __syncthreads();
      // Process the loaded balls.
      for (std::size_t i = 0; i != block; ++i) {
         // This accounts for about a third of the execution time.
         d = (p.x - b[i].x) * (p.x - b[i].x) + (p.y - b[i].y) * (p.y - b[i].y) +
            (p.z - b[i].z) * (p.z - b[i].z) - b[i].w;
         if (d < g) {
            g = d;
         }
      }
      ballIndices += block;
      numInfluencingBalls -= block;
   }
   // Record the patch value. 
   // Note: This statement accounts for about 35% of execution time.
   patch[tid] = g;
#endif
}


// Select a single patch using the block index. Then call a kernel for that
// patch.
__global__
void
powerDistanceKernel(const std::size_t numRefined,
                    float* patches,
                    const uint3* indices,
                    const float3 lowerCorner,
                    const float spacing,
                    const float4* ballsSquared,
                    const std::size_t* ballIndexOffsets,
                    const std::size_t* packedBallIndices) {
   // Convert the 2-D block index into a single patch index.
   const std::size_t i = blockIdx.x + blockIdx.y * gridDim.x;
   if (i >= numRefined) {
      return;
   }
   const std::size_t begin = ballIndexOffsets[i];
   const std::size_t end = ballIndexOffsets[i+1];
   const std::size_t NumThreads = PatchExtent * PatchExtent * PatchExtent;
   const float3 patchLowerCorner =
      {lowerCorner.x + PatchExtent * indices[i].x * spacing,
       lowerCorner.y + PatchExtent * indices[i].y * spacing,
       lowerCorner.z + PatchExtent * indices[i].z * spacing};
   powerDistanceKernel(patches + i * NumThreads,
                       patchLowerCorner, spacing, ballsSquared,
                       end - begin, &packedBallIndices[begin]);
}


#ifndef __CUDA_ARCH__
// Construct a level set for the power distance to a set of balls.
void
negativePowerDistanceCuda(Grid<float, 3, PatchExtent>* grid,
                          const std::vector<geom::Ball<float, 3> >& balls) {
   // Dispense with the trivial case.
   if (grid->empty()) {
      return;
   }

   // Determine the patch/ball dependencies.
   array::StaticArrayOfArrays<std::size_t> dependencies;
   {
      // 1.1 * (diagonal length of a voxel)
      const float offset = 1.1 * grid->spacing * std::sqrt(float(3));
      // Offset the ball radii to include the volume of calculated distance.
      std::vector<geom::Ball<float, 3> > offsetBalls(balls);
      for (std::size_t i = 0; i != offsetBalls.size(); ++i) {
         offsetBalls[i].radius += offset;
      }
      // Calculate the dependencies.
      patchDependencies(*grid, offsetBalls.begin(), offsetBalls.end(),
                        &dependencies);
   }

   // Refine the appropriate patches and set the rest to have an unknown
   // distance.
   grid->refine(dependencies);
   const std::size_t numRefined = grid->numRefined();

   // Allocate device memory for the refined patches and their indices.
   float* patchesDev;
   uint3* indicesDev;
   allocateGridCuda(*grid, numRefined, &patchesDev, &indicesDev);

   // Allocate device memory for the balls and copy the memory.
   float4* ballsSquaredDev;
   CUDA_CHECK(hipMalloc((void**)&ballsSquaredDev,
                         balls.size() * sizeof(float4)));
   {
      std::vector<float4> buffer(balls.size());
      for (std::size_t i = 0; i != balls.size(); ++i) {
         buffer[i].x = balls[i].center[0];
         buffer[i].y = balls[i].center[1];
         buffer[i].z = balls[i].center[2];
         buffer[i].w = balls[i].radius * balls[i].radius;
      }
      CUDA_CHECK(hipMemcpy(ballsSquaredDev, &buffer[0],
                            buffer.size() * sizeof(float4),
                            hipMemcpyHostToDevice));
   }

   // Allocate device memory for the ball index offsets and packed ball indices.
   // Copy the data to the device.
   std::size_t* ballIndexOffsetsDev;
   std::size_t* packedBallIndicesDev;
   allocateBallIndicesCuda(dependencies, numRefined, &ballIndexOffsetsDev,
                           &packedBallIndicesDev);
   
   const float3 lowerCorner = {grid->lowerCorner[0], grid->lowerCorner[1],
                               grid->lowerCorner[2]};
   // Use a 2-D grid of blocks. Because the number of refined patches may 
   // exceed the maximum allowed single grid dimension.
   hipDeviceProp_t prop;
   CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
   const std::size_t Len = prop.maxGridSize[0];
   const dim3 GridDim(std::min(numRefined, Len), (numRefined + Len - 1) / Len);
   // A thread for each patch grid point.
   const dim3 ThreadsPerBlock(PatchExtent, PatchExtent, PatchExtent);
   // Launch the kernel.
   powerDistanceKernel<<<GridDim,ThreadsPerBlock>>>
      (numRefined, patchesDev, indicesDev, lowerCorner, grid->spacing,
       ballsSquaredDev, ballIndexOffsetsDev, packedBallIndicesDev);
   // Copy the patch data back to the host.
   CUDA_CHECK(hipMemcpy(grid->data(), patchesDev,
                         grid->numVertices() * sizeof(float),
                         hipMemcpyDeviceToHost));

   // Free the device memory.
   CUDA_CHECK(hipFree(patchesDev));
   CUDA_CHECK(hipFree(indicesDev));
   CUDA_CHECK(hipFree(ballsSquaredDev));
   CUDA_CHECK(hipFree(ballIndexOffsetsDev));
   CUDA_CHECK(hipFree(packedBallIndicesDev));
}
#endif

} // namespace levelSet
